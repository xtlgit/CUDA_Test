
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {
    printf("Hello from GPU thread %d\n", threadIdx.x);
}

int main() {
    hello<<<1, 5>>>();
    hipDeviceSynchronize();
    printf("Hello from CPU\n");
    return 0;
}